
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("hello world form the GPU\n");
}

int main(void)
{
    hello_from_gpu<<<2, 4>>>(); //2個線程塊、每個線程塊有4個線程，共輸出8次
    hipDeviceSynchronize();

    return 0;
}