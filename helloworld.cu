
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("hello world form the GPU\n");
}

int main(void)
{
    hello_from_gpu<<<1, 1>>>();
    hipDeviceSynchronize();

    return 0;
}